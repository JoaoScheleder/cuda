#include  "hip/hip_runtime.h"
#include <stdio.h>

// malloc == cudaMalloc // Allocate device memory
// free == cudaFree // Free device memory
// memcpy == cudaMemcpy // Send from host (RAM) to device (GPU memory)

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = 10 * sizeof(int); // bytes

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 5;
    b = 7;


    // CPU -> GPU
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);


    add<<<1,1>>>(d_a, d_b, d_c);

    // GPU -> CPU
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost); // Already calls cudaDeviceSynchronize() internally
    
    printf("Result of %d + %d = %d \n", a, b, c);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    
    return 0;
}