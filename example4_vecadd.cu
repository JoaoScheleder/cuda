#include "hip/hip_runtime.h"
#include <stdio.h>

// When N > Total number of threads, each thread processes multiple elements
__global__ void vecAdd(const int *d_a, const int *d_b, int *d_c, int N)
{
    // int idx = threadIdx.x; // Local Index, could lead to multiple threads processing the same element if N > blockDim.x
    // int stride = blockDim.x; // Local Stride, could lead to multiple threads processing the same element if N > blockDim.x

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // índice global // Avoid using global idx directly to prevent out-of-bounds access
    int stride = blockDim.x * gridDim.x;             // salto global entre threads // Total number of threads in the grid

    // idx 0 will process elements 0, stride, 2*stride, ...
    for (int i = idx; i < N; i += stride)
    {
        printf("Thread ID: %d, Stride: %d\n, Processing element: %d\n", idx, stride, i);
        // if(i == 0){
        //     printf("Thread ID: %d, Stride: %d\n, Processing element: %d\n", idx, stride, i);
        // }
        d_c[i] = d_a[i] + d_b[i];
    }
}

// When N <= Total number of threads, each thread processes one element
__global__ void simpleVecAdd(const int *d_a, const int *d_b, int *d_c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // índice global
    d_c[idx] = d_a[idx] + d_b[idx];
}

// Same as the generic vecAdd kernel but with an initial if condition to handle the first element, same speed as the generic one
__global__ void vecAddHybrid(const int *d_a, const int *d_b, int *d_c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (idx < N)
    {
        d_c[idx] = d_a[idx] + d_b[idx];
    }

    for (int i = idx + stride; i < N; i += stride)
    {
        d_c[i] = d_a[i] + d_b[i];
    }
}

// vec add and while for single SM (Streaming Multiprocessor), in case it have more SM's it will not work properly.
// because blockIdx.x is considered in only one block (1 SM).
__global__ void vecAddWhile(const int *d_a, const int *d_b, int *d_c, int N)
{
    int idx = threadIdx.x;
    int i = idx;
    while (i < N)
    {
        d_c[i] = d_a[i] + d_b[i];
        i += blockDim.x;
    }
}

int main()
{
    hipDeviceReset();
    int N = 64;
    const int size = N * sizeof(int);

    int threadsPerBlock = 32;
    int blocksPerGrid = 1;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    printf("Total threads: %d\n", blocksPerGrid * threadsPerBlock);
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = i;
    }

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    printf("h_a[10]: %d\n", h_a[63]);
    printf("h_b[10]: %d\n", h_b[63]);
    printf("Result: %d\n", h_c[63]);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}