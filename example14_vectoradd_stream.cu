#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void vectorAddShared(const float *A, const float *B, float *C, int N)
{
    extern __shared__ float sharedMem[];
    float *sA = sharedMem;
    float *sB = &sharedMem[blockDim.x];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (idx < N) {
        sA[threadIdx.x] = A[idx];
        sB[threadIdx.x] = B[idx];
    } else {
        sA[threadIdx.x] = 0.0f;
        sB[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Perform vector addition
    if (idx < N) {
        C[idx] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}

int main () {
    const int N = 1 << 20; // 1M elements
    size_t size = N * sizeof(float);

    
    float *h_a, *h_b, *h_c;

    float *d_a0, *d_b0, *d_c0;
    float *d_a1, *d_b1, *d_c1;

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    hipHostAlloc((void**)&h_a, size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_c, size, hipHostMallocDefault);

    for (int i = 0; i < N; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    hipMalloc((void**)&d_a0, size);
    hipMalloc((void**)&d_b0, size);
    hipMalloc((void**)&d_c0, size);

    hipMalloc((void**)&d_a1, size);
    hipMalloc((void**)&d_b1, size);
    hipMalloc((void**)&d_c1, size);

    int blockSize = 256;
    int gridSize = ceil((float)N / blockSize);

    int sharedMemSize = 2 * blockSize * sizeof(float);

    hipMemcpyAsync(d_a0, h_a, size, hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(d_b0, h_b, size, hipMemcpyHostToDevice, stream0);
    vectorAddShared<<<gridSize, blockSize, sharedMemSize, stream0>>>(d_a0, d_b0, d_c0, N);
    
    hipMemcpyAsync(h_c, d_c0, size, hipMemcpyDeviceToHost, stream0);
    
    hipMemcpyAsync(d_a1, h_a, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_b1, h_b, size, hipMemcpyHostToDevice, stream1);
    vectorAddShared<<<gridSize, blockSize, sharedMemSize, stream1>>>(d_a1, d_b1, d_c1, N);
    
    hipMemcpyAsync(h_c, d_c1, size, hipMemcpyDeviceToHost, stream1);


    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);


    hipFree(d_a0);
    hipFree(d_b0);
    hipFree(d_c0);

    hipFree(d_a1);
    hipFree(d_b1);
    hipFree(d_c1);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
  
    return 0;  
};
